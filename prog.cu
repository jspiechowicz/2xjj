/*
 * Two identical overdamped AC-driven Josephson junctions coupled by external resistive shunt
 *
 * \dot{\phi}_1 = - sin(\phi_1) + i_1(t) + \alpha[I_2(t) - sin(\phi_2)] + \sqrt{Dg}\eta_1(t)
 * \dot{\phi}_2 = - sin(\phi_2) + i_2(t) + \alpha[I_1(t) - sin(\phi_1)] + \sqrt{Dg}\eta_2(t)
 *
 * see J. Spiechowicz et al., Acta Phys. Polon. B 43, 1203 (2012)
 *
 */

#include <stdio.h>
#include <getopt.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define PI 3.14159265358979f

//model
__constant__ float d_amp, d_omega, d_force, d_alpha, d_Dg, d_Dp, d_lambda;
__constant__ int d_comp;
float h_omega;

//simulation
float h_trans;
int h_dev, h_block, h_grid, h_spp;
long h_paths, h_periods, h_threads, h_steps, h_trigger;
__constant__ int d_spp, d_2ndorder;
__constant__ long d_paths, d_steps, d_trigger;

//output
char *h_domain;
char h_domainx, h_domainy;
float h_beginx, h_endx, h_beginy, h_endy;
int h_logx, h_logy, h_points, h_moments, h_traj, h_hist;
__constant__ char d_domainx;
__constant__ int d_points;

//vector
float *h_x1, *h_x2, *h_w, *h_x1b, *h_x2b, *h_dx;
float *d_x1, *d_x2, *d_w, *d_x1b, *d_x2b, *d_dx;
unsigned int *h_seeds, *d_seeds;
hiprandState *d_states;

size_t size_f, size_ui, size_p;
hiprandGenerator_t gen;

static struct option options[] = {
    {"amp", required_argument, NULL, 'a'},
    {"omega", required_argument, NULL, 'b'},
    {"force", required_argument, NULL, 'c'},
    {"alpha", required_argument, NULL, 'd'},
    {"Dg", required_argument, NULL, 'e'},
    {"Dp", required_argument, NULL, 'f'},
    {"lambda", required_argument, NULL, 'g'},
    {"comp", required_argument, NULL, 'h'},
    {"dev", required_argument, NULL, 'i'},
    {"block", required_argument, NULL, 'j'},
    {"paths", required_argument, NULL, 'k'},
    {"periods", required_argument, NULL, 'l'},
    {"trans", required_argument, NULL, 'm'},
    {"spp", required_argument, NULL, 'n'},
    {"algorithm", required_argument, NULL, 'o'},
    {"mode", required_argument, NULL, 'p'},
    {"domain", required_argument, NULL, 'q'},
    {"domainx", required_argument, NULL, 'r'},
    {"domainy", required_argument, NULL, 's'},
    {"logx", required_argument, NULL, 't'},
    {"logy", required_argument, NULL, 'u'},
    {"points", required_argument, NULL, 'v'},
    {"beginx", required_argument, NULL, 'w'},
    {"endx", required_argument, NULL, 'y'},
    {"beginy", required_argument, NULL, 'z'},
    {"endy", required_argument, NULL, 'A'}
};

void usage(char **argv)
{
    printf("Usage: %s <params> \n\n", argv[0]);
    printf("Model params:\n");
    printf("    -a, --amp=FLOAT         set the AC driving amplitude 'amp' to FLOAT\n");
    printf("    -b, --omega=FLOAT       set the AC driving frequency '\\omega' to FLOAT\n");
    printf("    -c, --force=FLOAT       set the external bias 'force' to FLOAT\n");
    printf("    -d, --alpha=FLOAT       set the coupling '\\alpha' to FLOAT\n");
    printf("    -e, --Dg=FLOAT          set the Gaussian noise intensity 'Dg' to FLOAT\n");
    printf("    -f, --Dp=FLOAT          set the Poissonian noise intensity 'Dp' to FLOAT\n");
    printf("    -g, --lambda=FLOAT      set the Poissonian kicks frequency '\\lambda' to FLOAT\n\n");
    printf("    -h, --comp=INT          choose between biased and unbiased Poissonian noise. INT can be one of:\n");
    printf("                            0: biased; 1: unbiased\n");
    printf("Simulation params:\n");
    printf("    -i, --dev=INT           set the gpu device to INT\n");
    printf("    -j, --block=INT         set the gpu block size to INT\n");
    printf("    -k, --paths=LONG        set the number of paths to LONG\n");
    printf("    -l, --periods=LONG      set the number of periods to LONG\n");
    printf("    -m, --trans=FLOAT       specify fraction FLOAT of periods which stands for transients\n");
    printf("    -n, --spp=INT           specify how many integration steps should be calculated\n");
    printf("                            for a single period of the driving force\n\n");
    printf("    -o, --algorithm=STRING  sets the algorithm. STRING can be one of:\n");
    printf("                            predcorr: simplified weak order 2.0 adapted predictor-corrector\n");
    printf("                            euler: simplified weak order 1.0 regular euler-maruyama\n");
    printf("Output params:\n");
    printf("    -p, --mode=STRING       sets the output mode. STRING can be one of:\n");
    printf("                            moments: the first moment <<v>>\n");
    printf("                            trajectory: ensemble averaged <x>(t)\n");
    printf("                            histogram: the final position of all paths\n");
    printf("    -q, --domain=STRING     simultaneously scan over one or two model params. STRING can be one of:\n");
    printf("                            1d: only one parameter; 2d: two parameters at once\n");
    printf("    -r, --domainx=CHAR      sets the first domain of the moments. CHAR can be one of:\n");
    printf("                            a: amp; w: omega, f: force; g: alpha; D: Dg; p: Dp; l: lambda\n");
    printf("    -s, --domainy=CHAR      sets the second domain of the moments (only if --domain=2d). CHAR can be the same as above.\n");
    printf("    -t, --logx=INT          choose between linear and logarithmic scale of the domainx\n");
    printf("                            0: linear; 1: logarithmic\n");
    printf("    -u, --logy=INT          the same as above but for domainy\n");
    printf("    -v, --points=INT        set the number of samples to generate between begin and end\n");
    printf("    -w, --beginx=FLOAT      set the starting value of the domainx to FLOAT\n");
    printf("    -y, --endx=FLOAT        set the end value of the domainx to FLOAT\n");
    printf("    -z, --beginy=FLOAT      the same as --beginx, but for domainy\n");
    printf("    -A, --endy=FLOAT        the same as --endx, but for domainy\n");
    printf("\n");
}

void parse_cla(int argc, char **argv)
{
    float ftmp;
    int c, itmp;

    while( (c = getopt_long(argc, argv, "a:b:c:d:e:f:g:h:i:j:k:l:m:n:o:p:q:r:s:t:u:v:w:y:z:A", options, NULL)) != EOF) {
        switch (c) {
            case 'a':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_amp), &ftmp, sizeof(float));
                break;
            case 'b':
                h_omega = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_omega), &h_omega, sizeof(float));
                break;
            case 'c':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_force), &ftmp, sizeof(float));
                break;
            case 'd':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_alpha), &ftmp, sizeof(float));
                break;
            case 'e':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_Dg), &ftmp, sizeof(float));
                break;
            case 'f':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_Dp), &ftmp, sizeof(float));
                break;
            case 'g':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_lambda), &ftmp, sizeof(float));
                break;
            case 'h':
                itmp = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_comp), &itmp, sizeof(int));
                break;
            case 'i':
                itmp = atoi(optarg);
                hipSetDevice(itmp);
                break;
            case 'j':
                h_block = atoi(optarg);
                break;
            case 'k':
                h_paths = atol(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_paths), &h_paths, sizeof(long));
                break;
            case 'l':
                h_periods = atol(optarg);
                break;
            case 'm':
                h_trans = atof(optarg);
                break;
            case 'n':
                h_spp = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_spp), &h_spp, sizeof(int));
                break;
            case 'o':
                if ( !strcmp(optarg, "predcorr") )
                    itmp = 1;
                else if ( !strcmp(optarg, "euler") )
                    itmp = 0;
                hipMemcpyToSymbol(HIP_SYMBOL(d_2ndorder), &itmp, sizeof(int));
                break;
            case 'p':
                if ( !strcmp(optarg, "moments") ) {
                    h_moments = 1;
                    h_traj = 0;
                    h_hist = 0;
                } else if ( !strcmp(optarg, "trajectory") ) {
                    h_traj = 1;
                    h_hist = 0;
                    h_moments = 0;
                } else if ( !strcmp(optarg, "histogram") ) {
                    h_moments = 0;
                    h_traj = 0;
                    h_hist = 1;
                }
                break;
            case 'q':
                h_domain = optarg;
                break;
            case 'r':
                h_domainx = optarg[0]; 
                hipMemcpyToSymbol(HIP_SYMBOL(d_domainx), &h_domainx, sizeof(char));
                break;
            case 's':
                h_domainy = optarg[0];
                break;
            case 't':
                h_logx = atoi(optarg);
                break;
            case 'u':
                h_logy = atoi(optarg);
                break;
            case 'v':
                h_points = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_points), &h_points, sizeof(int));
                break;
            case 'w':
                h_beginx = atof(optarg);
                break;
            case 'y':
                h_endx = atof(optarg);
                break;
            case 'z':
                h_beginy = atof(optarg);
                break;
            case 'A':
                h_endy = atof(optarg);
                break;
        }
    }
}

__global__ void init_dev_rng(unsigned int *d_seeds, hiprandState *d_states)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(d_seeds[idx], idx, 0, &d_states[idx]);
}

__device__ float drift(float l_x, float l_w, float l_amp, float l_force)
{
    return -sinf(l_x) + l_amp*cosf(l_w) + l_force;
}

__device__ float diffusion(float l_Dg, float l_dt, int l_2ndorder, hiprandState *l_state)
{
    if (l_Dg != 0.0f) {
        float r = hiprand_uniform(l_state);
        if (l_2ndorder) {
            if ( r <= 1.0f/6 ) {
                return -sqrtf(3.0f*l_Dg*l_dt);
            } else if ( r > 1.0f/6 && r <= 2.0f/6 ) {
                return sqrtf(3.0f*l_Dg*l_dt);
            } else {
                return 0.0f;
            }
        } else {
            if ( r <= 0.5f ) {
                return -sqrtf(l_Dg*l_dt);
            } else {
                return sqrtf(l_Dg*l_dt);
            }
        }
    } else {
        return 0.0f;
    }
}

__device__ float adapted_jump(int &npcd, int pcd, float l_lambda, float l_Dp, int l_comp, float l_dt, hiprandState *l_state)
{
    if (l_Dp != 0.0f) {
        float comp = sqrtf(l_Dp*l_lambda)*l_dt;
        if (pcd <= 0) {
            float ampmean = sqrtf(l_lambda/l_Dp);
           
            npcd = (int) floor( -logf( hiprand_uniform(l_state) )/l_lambda/l_dt + 0.5f );

            if (l_comp) {
                return -logf( hiprand_uniform(l_state) )/ampmean - comp;
            } else {
                return -logf( hiprand_uniform(l_state) )/ampmean;
            }
        } else {
            npcd = pcd - 1;
            if (l_comp) {
                return -comp;
            } else {
                return 0.0f;
            }
        }
    } else {
        return 0.0f;
    }
}

__device__ float regular_jump(float l_lambda, float l_Dp, int l_comp, float l_dt, hiprandState *l_state)
{
    if (l_Dp != 0.0f) {
        float mu, ampmean, comp, s;
        int i;
        unsigned int n;

        mu = l_lambda*l_dt;
        ampmean = sqrtf(l_lambda/l_Dp);
        comp = sqrtf(l_Dp*l_lambda)*l_dt;
        n = hiprand_poisson(l_state, mu);
        s = 0.0f;
            for (i = 0; i < n; i++) {
                s += -logf( hiprand_uniform(l_state) )/ampmean;
            }
        if (l_comp) s -= comp;
        return s;
    } else {
        return 0.0f;
    }
}

__device__ void predcorr(float &corrl_x1, float l_x1, float &corrl_x2, float l_x2, float &corrl_w, float l_w, int &npcd, int pcd, hiprandState *l_state, \
                         float l_amp, float l_omega, float l_force, float l_alpha, float l_Dg, int l_2ndorder, float l_Dp, float l_lambda, int l_comp, float l_dt)
/* simplified weak order 2.0 adapted predictor-corrector scheme
( see E. Platen, N. Bruti-Liberati; Numerical Solution of Stochastic Differential Equations with Jumps in Finance; Springer 2010; p. 503, p. 532 )
*/
{
    float l_x1t, l_x1tt, l_x2t, l_x2tt, l_wt, l_wtt, predl_x1, predl_x2, predl_w;

    l_x1t = drift(l_x1, l_w, l_amp, l_force) - l_alpha*sinf(l_x2);
    l_x2t = -sinf(l_x2) + l_alpha*drift(l_x1, l_w, l_amp, l_force);
    l_wt = l_omega;

    predl_x1 = l_x1 + l_x1t*l_dt + diffusion(l_Dg, l_dt, l_2ndorder, l_state);
    predl_x2 = l_x2 + l_x2t*l_dt + diffusion(l_Dg, l_dt, l_2ndorder, l_state);
    predl_w = l_w + l_wt*l_dt;

    l_x1tt = drift(predl_x1, predl_w, l_amp, l_force) - l_alpha*sinf(predl_x2);
    l_x2tt = -sinf(predl_x2) + l_alpha*drift(predl_x1, predl_w, l_amp, l_force);
    l_wtt = l_omega;

    predl_x1 = l_x1 + 0.5f*(l_x1t + l_x1tt)*l_dt + diffusion(l_Dg, l_dt, l_2ndorder, l_state);
    predl_x2 = l_x2 + 0.5f*(l_x2t + l_x2tt)*l_dt + diffusion(l_Dg, l_dt, l_2ndorder, l_state);
    predl_w = l_w + 0.5f*(l_wt + l_wtt)*l_dt;

    l_x1tt = drift(predl_x1, predl_w, l_amp, l_force) - l_alpha*sinf(predl_x2);
    l_x2tt = -sinf(predl_x2) + l_alpha*drift(predl_x1, predl_w, l_amp, l_force);
    l_wtt = l_omega;

    corrl_x1 = l_x1 + 0.5f*(l_x1t + l_x1tt)*l_dt + diffusion(l_Dg, l_dt, l_2ndorder, l_state) + adapted_jump(npcd, pcd, l_lambda, l_Dp, l_comp, l_dt, l_state);
    corrl_x2 = l_x2 + 0.5f*(l_x2t + l_x2tt)*l_dt + diffusion(l_Dg, l_dt, l_2ndorder, l_state);
    corrl_w = l_w + 0.5f*(l_wt + l_wtt)*l_dt;
}

__device__ void eulermaruyama(float &nl_x1, float l_x1, float &nl_x2, float l_x2, float &nl_w, float l_w, hiprandState *l_state, \
                         float l_amp, float l_omega, float l_force, float l_alpha, float l_Dg, int l_2ndorder, float l_Dp, float l_lambda, int l_comp, float l_dt)
/* simplified weak order 1.0 regular euler-maruyama scheme 
( see E. Platen, N. Bruti-Liberati; Numerical Solution of Stochastic Differential Equations with Jumps in Finance; Springer 2010; p. 508, 
  C. Kim, E. Lee, P. Talkner, and P.Hanggi; Phys. Rev. E 76; 011109; 2007 ) 
*/ 
{
    float l_x1t, l_x2t, l_wt;

    l_x1t = l_x1 + ( drift(l_x1, l_w, l_amp, l_force) - l_alpha*sinf(l_x2) )*l_dt + diffusion(l_Dg, l_dt, l_2ndorder, l_state) 
                 + regular_jump(l_lambda, l_Dp, l_comp, l_dt, l_state);
    l_x2t = l_x2 - sinf(l_x2)*l_dt + l_alpha*( drift(l_x1, l_w, l_amp, l_force)*l_dt + regular_jump(l_lambda, l_Dp, l_comp, l_dt, l_state) )
                 + diffusion(l_Dg, l_dt, l_2ndorder, l_state);
    l_wt = l_w + l_omega*l_dt;

    nl_x1 = l_x1t;
    nl_x2 = l_x2t;
    nl_w = l_wt;
}

__device__ void fold(float &nx, float x, float y, float &nfc, float fc)
//reduce periodic variable to the base domain
{
    nx = x - floor(x/y)*y;
    nfc = fc + floor(x/y)*y;
}

__global__ void run_moments(float *d_x1, float *d_x2, float *d_w, float *d_x1b, float *d_x2b, float *d_dx, hiprandState *d_states)
//actual moments kernel
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    float l_x1, l_x2, l_w, l_x1b, l_x2b, l_dx; 
    hiprandState l_state;

    //cache path and model parameters in local variables
    l_x1 = d_x1[idx];
    l_x2 = d_x2[idx];
    l_w = d_w[idx];
    l_x1b = d_x1b[idx];
    l_x2b = d_x2b[idx];
    l_state = d_states[idx];

    float l_amp, l_omega, l_force, l_alpha, l_Dg, l_Dp, l_lambda;
    int l_comp;

    l_amp = d_amp;
    l_omega = d_omega;
    l_force = d_force;
    l_alpha = d_alpha;
    l_Dg = d_Dg;
    l_Dp = d_Dp;
    l_lambda = d_lambda;
    l_comp = d_comp;

    //run simulation for multiple values of the system parameters
    long ridx = (idx/d_paths) % d_points;
    l_dx = d_dx[ridx];

    switch(d_domainx) {
        case 'a':
            l_amp = l_dx;
            break;
        case 'w':
            l_omega = l_dx;
            break;
        case 'f':
            l_force = l_dx;
            break;
        case 'g':
            l_alpha = l_dx;
            break;
        case 'D':
            l_Dg = l_dx;
            break;
        case 'p':
            l_Dp = l_dx;
            break;
        case 'l':
            l_lambda = l_dx;
            break;
    }

    //step size & number of steps
    float l_dt;
    long l_steps, l_trigger, i;

    l_dt = 2.0f*PI/l_omega/d_spp; 
    l_steps = d_steps;
    l_trigger = d_trigger;

    //counters for folding
    float x1fc, x2fc, wfc;
    
    x1fc = 0.0f;
    x2fc = 0.0f;
    wfc = 0.0f;

    int l_2ndorder, pcd;

    l_2ndorder = d_2ndorder;

    if (l_2ndorder) {
        //jump countdown
        pcd = (int) floor( -logf( hiprand_uniform(&l_state) )/l_lambda/l_dt + 0.5f );
    }
    
    for (i = 0; i < l_steps; i++) {

        //algorithm
        if (l_2ndorder) {
            predcorr(l_x1, l_x1, l_x2, l_x2, l_w, l_w, pcd, pcd, &l_state, l_amp, l_omega, l_force, l_alpha, l_Dg, l_2ndorder, l_Dp, l_lambda, l_comp, l_dt);
        } else {
            eulermaruyama(l_x1, l_x1, l_x2, l_x2, l_w, l_w, &l_state, l_amp, l_omega, l_force, l_alpha, l_Dg, l_2ndorder, l_Dp, l_lambda, l_comp, l_dt);
        }
        
        //fold path parameters
        if ( fabs(l_x1) > (2.0f*PI) ) {
            fold(l_x1, l_x1, (2.0f*PI), x1fc, x1fc);
        }

        if ( fabs(l_x2) > (2.0f*PI) ) {
            fold(l_x2, l_x2, (2.0f*PI), x2fc, x2fc);
        }

        if ( l_w > (2.0f*PI) ) {
            fold(l_w, l_w, (2.0f*PI), wfc, wfc);
        }

        if (i == l_trigger) {
            l_x1b = l_x1 + x1fc;
            l_x2b = l_x2 + x2fc;
        }

    }

    //write back path parameters to the global memory
    d_x1[idx] = l_x1 + x1fc;
    d_x2[idx] = l_x2 + x2fc;
    d_w[idx] = l_w;
    d_x1b[idx] = l_x1b;
    d_x2b[idx] = l_x2b;
    d_states[idx] = l_state;
}

__global__ void run_traj(float *d_x1, float *d_x2, float *d_w, hiprandState *d_states)
//actual trajectory kernel
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    float l_x1, l_x2, l_w; 
    hiprandState l_state;

    //cache path and model parameters in local variables
    l_x1 = d_x1[idx];
    l_x2 = d_x2[idx];
    l_w = d_w[idx];
    l_state = d_states[idx];

    float l_amp, l_omega, l_force, l_alpha, l_Dg, l_Dp, l_lambda;
    int l_comp;

    l_amp = d_amp;
    l_omega = d_omega;
    l_force = d_force;
    l_alpha = d_alpha;
    l_Dg = d_Dg;
    l_Dp = d_Dp;
    l_lambda = d_lambda;
    l_comp = d_comp;

    //step size & number of steps
    float l_dt;
    long l_steps, i;

    l_dt = 2.0f*PI/l_omega/d_spp; 
    l_steps = d_steps;

    //counters for folding
    float x1fc, x2fc, wfc;
    
    x1fc = 0.0f;
    x2fc = 0.0f;
    wfc = 0.0f;

    int l_2ndorder, pcd;

    l_2ndorder = d_2ndorder;

    if (l_2ndorder) {
        //jump countdown
        pcd = (int) floor( -logf( hiprand_uniform(&l_state) )/l_lambda/l_dt + 0.5f );
    }
    
    for (i = 0; i < l_steps; i++) {

        //algorithm
        if (l_2ndorder) {
            predcorr(l_x1, l_x1, l_x2, l_x2, l_w, l_w, pcd, pcd, &l_state, l_amp, l_omega, l_force, l_alpha, l_Dg, l_2ndorder, l_Dp, l_lambda, l_comp, l_dt);
        } else {
            eulermaruyama(l_x1, l_x1, l_x2, l_x2, l_w, l_w, &l_state, l_amp, l_omega, l_force, l_alpha, l_Dg, l_2ndorder, l_Dp, l_lambda, l_comp, l_dt);
        }
        
        //fold path parameters
        if ( fabs(l_x1) > (2.0f*PI) ) {
            fold(l_x1, l_x1, (2.0f*PI), x1fc, x1fc);
        }

        if ( fabs(l_x2) > (2.0f*PI) ) {
            fold(l_x2, l_x2, (2.0f*PI), x2fc, x2fc);
        }

        if ( l_w > (2.0f*PI) ) {
            fold(l_w, l_w, (2.0f*PI), wfc, wfc);
        }

    }

    //write back path parameters to the global memory
    d_x1[idx] = l_x1 + x1fc;
    d_x2[idx] = l_x2 + x2fc;
    d_w[idx] = l_w;
    d_states[idx] = l_state;
}

void prepare()
//prepare simulation
{
    //grid size
    h_paths = (h_paths/h_block)*h_block;
    h_threads = h_paths;

    if (h_moments) h_threads *= h_points;

    h_grid = h_threads/h_block;

    //number of steps
    if (h_traj) {
        h_steps = h_spp;
    } else {
        h_steps = h_periods*h_spp;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(d_steps), &h_steps, sizeof(long));
     
    //host memory allocation
    size_f = h_threads*sizeof(float);
    size_ui = h_threads*sizeof(unsigned int);
    size_p = h_points*sizeof(float);

    h_x1 = (float*)malloc(size_f);
    h_x2 = (float*)malloc(size_f);
    h_w = (float*)malloc(size_f);
    h_seeds = (unsigned int*)malloc(size_ui);

    //create & initialize host rng
    hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

    hiprandGenerate(gen, h_seeds, h_threads);
 
    //device memory allocation
    hipMalloc((void**)&d_x1, size_f);
    hipMalloc((void**)&d_x2, size_f);
    hipMalloc((void**)&d_w, size_f);
    hipMalloc((void**)&d_seeds, size_ui);
    hipMalloc((void**)&d_states, h_threads*sizeof(hiprandState));

    //copy seeds from host to device
    hipMemcpy(d_seeds, h_seeds, size_ui, hipMemcpyHostToDevice);

    //initialization of device rng
    init_dev_rng<<<h_grid, h_block>>>(d_seeds, d_states);

    free(h_seeds);
    hipFree(d_seeds);

    //moments specific requirements
    if (h_moments) {
        h_trigger = h_steps*h_trans;
        hipMemcpyToSymbol(HIP_SYMBOL(d_trigger), &h_trigger, sizeof(long));

        h_x1b = (float*)malloc(size_f);
        h_x2b = (float*)malloc(size_f);
        h_dx = (float*)malloc(size_p);

        float dxtmp = h_beginx;
        float dxstep = (h_endx - h_beginx)/h_points;

        long i;
        
        //set domainx
        for (i = 0; i < h_points; i++) {
            if (h_logx) {
                h_dx[i] = pow(10.0f, dxtmp);
            } else {
                h_dx[i] = dxtmp;
            }
            dxtmp += dxstep;
        }
        
        hipMalloc((void**)&d_x1b, size_f);
        hipMalloc((void**)&d_x2b, size_f);
        hipMalloc((void**)&d_dx, size_p);
    
        hipMemcpy(d_dx, h_dx, size_p, hipMemcpyHostToDevice);
    }
}

void copy_to_dev()
{
    hipMemcpy(d_x1, h_x1, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_x2, h_x2, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w, size_f, hipMemcpyHostToDevice);
    if (h_moments) {
        hipMemcpy(d_x1b, h_x1b, size_f, hipMemcpyHostToDevice);
        hipMemcpy(d_x2b, h_x2b, size_f, hipMemcpyHostToDevice);
    }
}

void copy_from_dev()
{
    hipMemcpy(h_x1, d_x1, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_x2, d_x2, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_w, d_w, size_f, hipMemcpyDeviceToHost);
    if (h_moments) {
        hipMemcpy(h_x1b, d_x1b, size_f, hipMemcpyDeviceToHost);
        hipMemcpy(h_x2b, d_x2b, size_f, hipMemcpyDeviceToHost);
    }
}

void initial_conditions()
//set initial conditions for path parameters
{
    hiprandGenerateUniform(gen, h_x1, h_threads);
    hiprandGenerateUniform(gen, h_x2, h_threads);
    hiprandGenerateUniform(gen, h_w, h_threads);

    long i;

    for (i = 0; i < h_threads; i++) {
        h_x1[i] *= 2.0f*PI; //x1 in (0,2\pi]
        h_x2[i] *= 2.0f*PI; //x2 in (0,2\pi]
        h_w[i] *= 2.0f*PI; //w in (0,2\pi]
    }

    if (h_moments) {
        memset(h_x1b, 0, size_f);
        memset(h_x2b, 0, size_f);
    }
    
    copy_to_dev();
}

void moments(float *av, float *av2)
//calculate the first two moments of v
{
    float sx1, sx1b, sx2, sx2b;
    int i, j;

    hipMemcpy(h_x1, d_x1, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_x1b, d_x1b, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_x2, d_x2, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_x2b, d_x2b, size_f, hipMemcpyDeviceToHost);

    for (j = 0; j < h_points; j++) {
        sx1 = 0.0f;
        sx1b = 0.0f;
        sx2 = 0.0f;
        sx2b = 0.0f;

        for (i = 0; i < h_paths; i++) {
            sx1 += h_x1[j*h_paths + i];
            sx1b += h_x1b[j*h_paths + i];
            sx2 += h_x2[j*h_paths + i];
            sx2b += h_x2b[j*h_paths + i];
        }

        av[j] = (sx1 - sx1b)/( (1.0f - h_trans)*h_periods*2.0f*PI/h_omega )/h_paths;
        av2[j] = (sx2 - sx2b)/( (1.0f - h_trans)*h_periods*2.0f*PI/h_omega )/h_paths;
    }
}

void ensemble_average(float *h_x1, float *h_x2, float &sx1, float &sx2)
//calculate ensemble average
{
    int i;

    sx1 = 0.0f;
    sx2 = 0.0f;

    for (i = 0; i < h_threads; i++) {
        sx1 += h_x1[i];
        sx2 += h_x2[i];
    }

    sx1 /= h_threads;
    sx2 /= h_threads;
}

void finish()
//free memory
{

    free(h_x1);
    free(h_x2);
    free(h_w);
    
    hiprandDestroyGenerator(gen);
    hipFree(d_x1);
    hipFree(d_x2);
    hipFree(d_w);
    hipFree(d_states);
    
    if (h_moments) {
        free(h_x1b);
        free(h_x2b);
        free(h_dx);

        hipFree(d_x1b);
        hipFree(d_x2b);
        hipFree(d_dx);
    }
}

int main(int argc, char **argv)
{
    parse_cla(argc, argv);
    if (!h_moments && !h_traj && !h_hist) {
        usage(argv);
        return -1;
    }

    prepare();
    
    initial_conditions();
    
    //asymptotic long time average velocity <<v>> and <<v^2>>
    if (h_moments) {
        float *av, *av2;
        int i;

        av = (float*)malloc(size_p);
        av2 = (float*)malloc(size_p);

        if ( !strcmp(h_domain, "1d") ) {
            run_moments<<<h_grid, h_block>>>(d_x1, d_x2, d_w, d_x1b, d_x2b, d_dx, d_states);
            moments(av, av2);

            printf("#%c <<v1>> <<v2>>\n", h_domainx);
            for (i = 0; i < h_points; i++) {
                printf("%e %e %e\n", h_dx[i], av[i], av2[i]);
            }

        } else {
            float h_dy, dytmp, dystep;
            int j;
            
            dytmp = h_beginy;
            dystep = (h_endy - h_beginy)/h_points;
            
            printf("#%c %c <<v1>> <<v2>>\n", h_domainx, h_domainy);
            
            for (i = 0; i < h_points; i++) {
                if (h_logy) {
                    h_dy = pow(10.0f, dytmp);
                } else {
                    h_dy = dytmp;
                }

                switch(h_domainy) {
                    case 'a':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_amp), &h_dy, sizeof(float));
                        break;
                    case 'w':
                        h_omega = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_omega), &h_omega, sizeof(float));
                        break;
                    case 'f':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_force), &h_dy, sizeof(float));
                        break;
                    case 'g':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_alpha), &h_dy, sizeof(float));
                        break;
                    case 'D':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_Dg), &h_dy, sizeof(float));
                        break;
                    case 'p':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_Dp), &h_dy, sizeof(float));
                        break;
                    case 'l':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_lambda), &h_dy, sizeof(float));
                        break;
                }

                run_moments<<<h_grid, h_block>>>(d_x1, d_x2, d_w, d_x1b, d_x2b, d_dx, d_states);
                moments(av, av2);
                
                for (j = 0; j < h_points; j++) {
                    printf("%e %e %e %e\n", h_dx[j], h_dy, av[j], av2[j]);
                }

                //blank line for plotting purposes
                printf("\n");

                initial_conditions();

                dytmp += dystep;
            }
        }

        free(av);
        free(av2);
    }

    //ensemble averaged trajectory <x>(t) 
    if (h_traj) {
        float t, sx1, sx2;
        int i;

        for (i = 0; i < h_periods; i++) {
            run_traj<<<h_grid, h_block>>>(d_x1, d_x2, d_w, d_states);
            copy_from_dev();
            t = i*2.0f*PI/h_omega;
            ensemble_average(h_x1, h_x2, sx1, sx2);
            printf("%e %e %e\n", t, sx1, sx2);
        }
    }

    //the final position of all paths
    if (h_hist) {
        int i;

        run_traj<<<h_grid, h_block>>>(d_x1, d_x2, d_w, d_states);
        copy_from_dev();
        
        for (i = 0; i < h_threads; i++) {
            printf("%e %e\n", h_x1[i], h_x2[i]);
        }
    }

    finish();

    return 0;
}
