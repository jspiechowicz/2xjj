#include "hip/hip_runtime.h"
/*
 * Two Underdamped Brownian Particles
 *
 */

#include <stdio.h>
#include <getopt.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define PI 3.14159265358979f

//model
__constant__ float d_amp, d_omega, d_force, d_gam, d_Dg;//, d_Dp, d_lambda, d_mean, d_fa, d_fb, d_mua, d_mub;
//__constant__ int d_comp;
float h_omega;//, h_lambda, h_fa, h_fb, h_mua, h_mub, h_mean;
//int h_comp;

//simulation
int h_dev, h_block, h_grid, h_spp, h_samples, h_2ndorder, h_trigger, h_paths, h_periods, h_trans;//, h_initnoise;
long h_threads, h_steps;
__constant__ int d_spp, d_2ndorder, d_samples, d_trigger, d_paths;//, d_initnoise;

//output
char *h_domain;
char h_domainx, h_domainy;
float h_beginx, h_endx, h_beginy, h_endy;
int h_logx, h_logy, h_points, h_moments, h_traj, h_hist;
__constant__ char d_domainx;
__constant__ int d_moments, d_points;

//vector
float *h_x1, *h_fx1, *h_v1, *h_x2, *h_fx2, *h_v2, *h_w, *h_fw, *h_sv1, *h_sv12, *h_sv2, *h_sv22, *h_dx;
float *d_x1, *d_fx1, *d_v1, *d_x2, *d_fx2, *d_v2, *d_w, *d_fw, *d_sv1, *d_sv12, *d_sv2, *d_sv22, *d_dx;
//int *d_pcd, *d_dcd, *d_dst;
unsigned int *h_seeds, *d_seeds;
hiprandState *d_states;

size_t size_f, size_i, size_ui, size_p;
hiprandGenerator_t gen;

//command line arguments
static struct option options[] = {
    {"amp", required_argument, NULL, 'a'},
    {"omega", required_argument, NULL, 'b'},
    {"force", required_argument, NULL, 'c'},
    {"gam", required_argument, NULL, 'd'},
    {"Dg", required_argument, NULL, 'e'},
/*    {"Dp", required_argument, NULL, 'f'},
    {"lambda", required_argument, NULL, 'g'},
    {"comp", required_argument, NULL, 'h'},*/
    {"dev", required_argument, NULL, 'i'},
    {"block", required_argument, NULL, 'j'},
    {"paths", required_argument, NULL, 'k'},
    {"periods", required_argument, NULL, 'l'},
    {"trans", required_argument, NULL, 'm'},
    {"spp", required_argument, NULL, 'n'},
    {"samples", required_argument, NULL, 'o'},
    {"algorithm", required_argument, NULL, 'p'},
    {"mode", required_argument, NULL, 'q'},
    {"domain", required_argument, NULL, 'r'},
    {"domainx", required_argument, NULL, 's'},
    {"domainy", required_argument, NULL, 't'},
    {"logx", required_argument, NULL, 'u'},
    {"logy", required_argument, NULL, 'v'},
    {"points", required_argument, NULL, 'w'},
    {"beginx", required_argument, NULL, 'y'},
    {"endx", required_argument, NULL, 'z'},
    {"beginy", required_argument, NULL, 'A'},
    {"endy", required_argument, NULL, 'B'}
/*    {"mean", required_argument, NULL, 'C'},
    {"fa", required_argument, NULL, 'D'},
    {"fb", required_argument, NULL, 'E'},
    {"mua", required_argument, NULL, 'F'},
    {"mub", required_argument, NULL, 'G'}*/
};

void usage(char **argv)
{
    printf("Usage: %s <params> \n\n", argv[0]);
    printf("Model params:\n");
    printf("    -a, --amp=FLOAT         set the harmonic driving amplitude 'a' to FLOAT\n");
    printf("    -b, --omega=FLOAT       set the harmonic driving frequency '\\omega' to FLOAT\n");
    printf("    -c, --force=FLOAT       set the external bias 'F' to FLOAT\n");
    printf("    -d, --gam=FLOAT         set the viscosity '\\gamma' to FLOAT\n");
    printf("    -e, --Dg=FLOAT          set the Gaussian noise intensity 'D_G' to FLOAT\n");
/*    printf("    -f, --Dp=FLOAT          set the Poissonian noise intensity 'D_P' to FLOAT\n");
    printf("    -g, --lambda=FLOAT      set the Poissonian kicks frequency '\\lambda' to FLOAT\n");
    printf("    -h, --comp=INT          choose between biased and unbiased Poissonian or dichotomous noise. INT can be one of:\n");
    printf("                            0: biased; 1: unbiased\n");
    printf("    -D, --fa=FLOAT          set the first state of the dichotomous noise 'F_a' to FLOAT\n");
    printf("    -E, --fb=FLOAT          set the second state of the dichotomous noise 'F_b' to FLOAT\n");
    printf("    -F, --mua=FLOAT         set the transition rate of the first state of dichotomous noise '\\mu_a' to FLOAT\n");
    printf("    -G, --mub=FLOAT         set the transition rate of the second state of dichotomous noise '\\mu_b' to FLOAT\n");
    printf("    -C, --mean=FLOAT        if is nonzero, fix the mean value of Poissonian noise or dichotomous noise to FLOAT, matters only for domains p, l, i, j, m or n\n");*/
    printf("Simulation params:\n");
    printf("    -i, --dev=INT           set the gpu device to INT\n");
    printf("    -j, --block=INT         set the gpu block size to INT\n");
    printf("    -k, --paths=INT         set the number of paths to INT\n");
    printf("    -l, --periods=INT       set the number of periods to INT\n");
    printf("    -m, --trans=FLOAT       specify fraction FLOAT of periods which stands for transients\n");
    printf("    -n, --spp=INT           specify how many integration steps should be calculated\n");
    printf("                            for a single period of the driving force\n");
    printf("    -o, --samples=INT       specify how many integration steps should be calculated for a single kernel call\n");
    printf("    -p, --algorithm=STRING  sets the algorithm. STRING can be one of:\n");
    printf("                            predcorr: simplified weak order 2.0 adapted predictor-corrector\n");
    printf("                            euler: simplified weak order 1.0 regular euler-maruyama\n");
    printf("Output params:\n");
    printf("    -q, --mode=STRING       sets the output mode. STRING can be one of:\n");
    printf("                            moments: the first two moments <<v>>, <<v^2>> and diffusion coefficient\n");
    printf("                            trajectory: ensemble averaged <x>(t), <v>(t) and <x^2>(t), <v^2>(t)\n");
    printf("                            histogram: the final position x and velocity v of all paths\n");
    printf("    -r, --domain=STRING     simultaneously scan over one or two model params. STRING can be one of:\n");
    printf("                            1d: only one parameter; 2d: two parameters at once\n");
    printf("    -s, --domainx=CHAR      sets the first domain of the moments. CHAR can be one of:\n");
    printf("                            a: amp; w: omega, f: force; g: gam; D: Dg\n");// p: Dp; l: lambda; i: fa; j: fb; m: mua; n: mub\n");
    printf("    -t, --domainy=CHAR      sets the second domain of the moments (only if --domain=2d). CHAR can be the same as above.\n");
    printf("    -u, --logx=INT          choose between linear and logarithmic scale of the domainx\n");
    printf("                            0: linear; 1: logarithmic\n");
    printf("    -v, --logy=INT          the same as above but for domainy\n");
    printf("    -w, --points=INT        set the number of samples to generate between begin and end\n");
    printf("    -y, --beginx=FLOAT      set the starting value of the domainx to FLOAT\n");
    printf("    -z, --endx=FLOAT        set the end value of the domainx to FLOAT\n");
    printf("    -A, --beginy=FLOAT      the same as --beginx, but for domainy\n");
    printf("    -B, --endy=FLOAT        the same as --endx, but for domainy\n");
    printf("\n");
}

//parse command line arguments
void parse_cla(int argc, char **argv)
{
    float ftmp;
    int c, itmp;

//    while( (c = getopt_long(argc, argv, "a:b:c:d:e:f:g:h:i:j:k:l:m:n:o:p:q:r:s:t:u:v:w:y:z:A:B:C:D:E:F:G", options, NULL)) != EOF) {
    while( (c = getopt_long(argc, argv, "a:b:c:d:e:i:j:k:l:m:n:o:p:q:r:s:t:u:v:w:y:z:A:B", options, NULL)) != EOF) {
        switch (c) {
            case 'a':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_amp), &ftmp, sizeof(float));
                break;
            case 'b':
                h_omega = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_omega), &h_omega, sizeof(float));
                break;
            case 'c':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_force), &ftmp, sizeof(float));
                break;
            case 'd':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_gam), &ftmp, sizeof(float));
                break;
            case 'e':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_Dg), &ftmp, sizeof(float));
                break;
/*            case 'f':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_Dp), &ftmp, sizeof(float));
                break;
            case 'g':
                h_lambda = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_lambda), &h_lambda, sizeof(float));
                break;
            case 'h':
                h_comp = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_comp), &h_comp, sizeof(int));
                break;*/
            case 'i':
                itmp = atoi(optarg);
                hipSetDevice(itmp);
                break;
            case 'j':
                h_block = atoi(optarg);
                break;
            case 'k':
                h_paths = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_paths), &h_paths, sizeof(int));
                break;
            case 'l':
                h_periods = atoi(optarg);
                break;
            case 'm':
                h_trans = atoi(optarg);
                break;
            case 'n':
                h_spp = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_spp), &h_spp, sizeof(int));
                break;
            case 'o':
                h_samples = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_samples), &h_samples, sizeof(int));
                break;
            case 'p':
                if ( !strcmp(optarg, "predcorr") )
                    h_2ndorder = 1;
                else if ( !strcmp(optarg, "euler") )
                    h_2ndorder = 0;
                hipMemcpyToSymbol(HIP_SYMBOL(d_2ndorder), &h_2ndorder, sizeof(int));
                break;
            case 'q':
                if ( !strcmp(optarg, "moments") ) {
                    h_moments = 1;
                    h_traj = 0;
                    h_hist = 0;
                } else if ( !strcmp(optarg, "trajectory") ) {
                    h_moments = 0;
                    h_traj = 1;
                    h_hist = 0;
                } else if ( !strcmp(optarg, "histogram") ) {
                    h_moments = 0;
                    h_traj = 0;
                    h_hist = 1;
                }
                hipMemcpyToSymbol(HIP_SYMBOL(d_moments), &h_moments, sizeof(int));
                break;
            case 'r':
                h_domain = optarg;
                break;
            case 's':
                h_domainx = optarg[0]; 
                hipMemcpyToSymbol(HIP_SYMBOL(d_domainx), &h_domainx, sizeof(char));
                break;
            case 't':
                h_domainy = optarg[0];
                break;
            case 'u':
                h_logx = atoi(optarg);
                break;
            case 'v':
                h_logy = atoi(optarg);
                break;
            case 'w':
                h_points = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_points), &h_points, sizeof(int));
                break;
            case 'y':
                h_beginx = atof(optarg);
                break;
            case 'z':
                h_endx = atof(optarg);
                break;
            case 'A':
                h_beginy = atof(optarg);
                break;
            case 'B':
                h_endy = atof(optarg);
                break;
/*            case 'C':
                h_mean = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_mean), &h_mean, sizeof(float));
                break;
            case 'D':
                h_fa = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_fa), &h_fa, sizeof(float));
                break;
            case 'E':
                h_fb = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_fb), &h_fb, sizeof(float));
                break;
            case 'F':
                h_mua = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_mua), &h_mua, sizeof(float));
                break;
            case 'G':
                h_mub = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_mub), &h_mub, sizeof(float));
                break;*/
        }
    }
}

//initialize device random number generator
__global__ void init_dev_rng(unsigned int *d_seeds, hiprandState *d_states)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(d_seeds[idx], 0, 0, &d_states[idx]);
}

__device__ float drift(float l_x, float l_v, float l_w, float l_gam, float l_amp, float l_force)
{
    return -l_gam*l_v - 2.0f*PI*cosf(2.0f*PI*l_x) + l_amp*cosf(l_w) + l_force;
}

__device__ float diffusion(float l_gam, float l_Dg, float l_dt, int l_2ndorder, hiprandState *l_state)
{
    if (l_Dg != 0.0f) {
        float r = hiprand_uniform(l_state);
        float g = sqrtf(2.0f*l_gam*l_Dg);
        if (l_2ndorder) {
            if ( r <= 1.0f/6 ) {
                return -g*sqrtf(3.0f*l_dt);
            } else if ( r > 1.0f/6 && r <= 2.0f/6 ) {
                return g*sqrtf(3.0f*l_dt);
            } else {
                return 0.0f;
            }
        } else {
            if ( r <= 0.5f ) {
                return -g*sqrtf(l_dt);
            } else {
                return g*sqrtf(l_dt);
            }
        }
    } else {
        return 0.0f;
    }
}

/*__device__ float adapted_jump_poisson(int &npcd, int pcd, float l_lambda, float l_Dp, int l_comp, float l_dt, hiprandState *l_state)
{
    if (l_lambda != 0.0f) {
        if (pcd <= 0) {
            float ampmean = sqrtf(l_lambda/l_Dp);
           
            npcd = (int) floorf( -logf( hiprand_uniform(l_state) )/l_lambda/l_dt + 0.5f );

            if (l_comp) {
                float comp = sqrtf(l_Dp*l_lambda)*l_dt;
                
                return -logf( hiprand_uniform(l_state) )/ampmean - comp;
            } else {
                return -logf( hiprand_uniform(l_state) )/ampmean;
            }
        } else {
            npcd = pcd - 1;
            if (l_comp) {
                float comp = sqrtf(l_Dp*l_lambda)*l_dt;
                
                return -comp;
            } else {
                return 0.0f;
            }
        }
    } else {
        return 0.0f;
    }
}

__device__ float adapted_jump_dich(int &ndcd, int dcd, int &ndst, int dst, float l_fa, float l_fb, float l_mua, float l_mub, float l_dt, hiprandState *l_state)
{
    if (l_mua != 0.0f || l_mub != 0.0f) {
        if (dcd <= 0) {
            if (dst == 0) {
                ndst = 1; 
                ndcd = (int) floorf( -logf( hiprand_uniform(l_state) )/l_mub/l_dt + 0.5f );
                return l_fb*l_dt;
            } else {
                ndst = 0;
                ndcd = (int) floorf( -logf( hiprand_uniform(l_state) )/l_mua/l_dt + 0.5f );
                return l_fa*l_dt;
            }
        } else {
            ndcd = dcd - 1;
            if (dst == 0) {
                return l_fa*l_dt;
            } else {
                return l_fb*l_dt;
            }
        }
    } else {
        return 0.0f;
    }
}

__device__ float regular_jump_poisson(float l_lambda, float l_Dp, int l_comp, float l_dt, hiprandState *l_state)
{
    if (l_lambda != 0.0f) {
        float mu, ampmean, comp, s;
        int i;
        unsigned int n;

        mu = l_lambda*l_dt;
        ampmean = sqrtf(l_lambda/l_Dp);
        comp = sqrtf(l_Dp*l_lambda)*l_dt;
        n = hiprand_poisson(l_state, mu);
        s = 0.0f;
            for (i = 0; i < n; i++) {
                s += -logf( hiprand_uniform(l_state) )/ampmean;
            }
        if (l_comp) s -= comp;
        return s;
    } else {
        return 0.0f;
    }
}*/

/* simplified weak order 2.0 adapted predictor-corrector scheme
( see E. Platen, N. Bruti-Liberati; Numerical Solution of Stochastic Differential Equations with Jumps in Finance; Springer 2010; p. 503, p. 532 )
*/
__device__ void predcorr(float &corrl_x, float l_x, float &corrl_v, float l_v, float &corrl_w, float l_w, int &npcd, int pcd, hiprandState *l_state, \
                         float l_amp, float l_omega, float l_force, float l_gam, float l_Dg, int l_2ndorder, float l_Dp, float l_lambda, int l_comp, \
                         int &ndcd, int dcd, int &ndst, int dst, float l_fa, float l_fb, float l_mua, float l_mub, float l_dt)
{
    float l_xt, l_xtt, l_vt, l_vtt, l_wt, l_wtt, predl_x, predl_v, predl_w;

    l_xt = l_v;
    l_vt = drift(l_x, l_v, l_w, l_gam, l_amp, l_force);
    l_wt = l_omega;

    predl_x = l_x + l_xt*l_dt;
    predl_v = l_v + l_vt*l_dt + diffusion(l_gam, l_Dg, l_dt, l_2ndorder, l_state);
    predl_w = l_w + l_wt*l_dt;

    l_xtt = predl_v;
    l_vtt = drift(predl_x, predl_v, predl_w, l_gam, l_amp, l_force);
    l_wtt = l_omega;

    predl_x = l_x + 0.5f*(l_xt + l_xtt)*l_dt;
    predl_v = l_v + 0.5f*(l_vt + l_vtt)*l_dt + diffusion(l_gam, l_Dg, l_dt, l_2ndorder, l_state);
    predl_w = l_w + 0.5f*(l_wt + l_wtt)*l_dt;

    l_xtt = predl_v;
    l_vtt = drift(predl_x, predl_v, predl_w, l_gam, l_amp, l_force);
    l_wtt = l_omega;

    corrl_x = l_x + 0.5f*(l_xt + l_xtt)*l_dt;
    corrl_v = l_v + 0.5f*(l_vt + l_vtt)*l_dt + diffusion(l_gam, l_Dg, l_dt, l_2ndorder, l_state) + adapted_jump_poisson(npcd, pcd, l_lambda, l_Dp, l_comp, l_dt, l_state) + adapted_jump_dich(ndcd, dcd, ndst, dst, l_fa, l_fb, l_mua, l_mub, l_dt, l_state);
    corrl_w = l_w + 0.5f*(l_wt + l_wtt)*l_dt;
}

/* simplified weak order 1.0 regular euler-maruyama scheme 
( see E. Platen, N. Bruti-Liberati; Numerical Solution of Stochastic Differential Equations with Jumps in Finance; Springer 2010; p. 508, 
  C. Kim, E. Lee, P. Talkner, and P.Hanggi; Phys. Rev. E 76; 011109; 2007 ) 
*/
__device__ void eulermaruyama(float &nl_x, float l_x, float &nl_v, float l_v, float &nl_w, float l_w, hiprandState *l_state, \
                         float l_amp, float l_omega, float l_force, float l_gam, float l_Dg, int l_2ndorder, float l_Dp, float l_lambda, int l_comp, \
                         int &ndcd, int dcd, int &ndst, int dst, float l_fa, float l_fb, float l_mua, float l_mub, float l_dt)
{
    float l_xt, l_vt, l_wt;

    l_vt = l_v + drift(l_x, l_v, l_w, l_gam, l_amp, l_force)*l_dt
               + diffusion(l_gam, l_Dg, l_dt, l_2ndorder, l_state)
               + regular_jump_poisson(l_lambda, l_Dp, l_comp, l_dt, l_state)
               + adapted_jump_dich(ndcd, dcd, ndst, dst, l_fa, l_fb, l_mua, l_mub, l_dt, l_state);
    l_xt = l_x + l_v*l_dt;
    l_wt = l_w + l_omega*l_dt;

    nl_v = l_vt;
    nl_x = l_xt;
    nl_w = l_wt;
}

//reduce periodic variable to the base domain
__global__ void fold(float *d_x, float *d_fx, float p)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    float l_x, l_fx, f;

    l_x = d_x[idx];
    l_fx = d_fx[idx];

    if (fabsf(l_x) >= p) {
        f = floorf(l_x/p)*p;
        l_x = l_x - f;
        l_fx = l_fx + f;
    }

    d_x[idx] = l_x;
    d_fx[idx] = l_fx;
}

//unfold periodic variable
void unfold(float *x, float *fx)
{
    long i;

    for (i = 0; i < h_threads; i++) {
        x[i] = x[i] + fx[i];
    }
}

//actual simulation kernel
__global__ void run_sim(float *d_x1, float *d_x2, float *d_v1, float *d_v2; float *d_w, float *d_sv1, float *d_sv2, float *d_sv12, float *d_sv22, float *d_dx, hiprandState *d_states); //int *d_pcd, int *d_dcd, int *d_dst, hiprandState *d_states)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //cache path and model parameters in local variables
    float l_x1, l_x2, l_v1, l_v2, l_w, l_sv1, l_sv2, l_sv12, l_sv22, l_dx; 
    hiprandState l_state;

    l_x1 = d_x1[idx];
    l_x2 = d_x2[idx];
    l_v1 = d_v1[idx];
    l_v2 = d_v2[idx];
    l_w = d_w[idx];
    l_sv1 = d_sv1[idx];
    l_sv2 = d_sv2[idx];
    l_sv12 = d_sv12[idx];
    l_sv22 = d_sv22[idx];
    l_state = d_states[idx];

    float l_amp, l_omega, l_force, l_gam, l_Dg;//, l_Dp, l_lambda, l_mean, l_fa, l_fb, l_mua, l_mub;
    int l_2ndorder;//, l_comp;
    int l_moments;

    l_amp = d_amp;
    l_omega = d_omega;
    l_force = d_force;
    l_gam = d_gam;
    l_Dg = d_Dg;
    l_Dp = d_Dp;
/*    l_lambda = d_lambda;
    l_mean = d_mean;
    l_comp = d_comp;
    l_fa = d_fa;
    l_fb = d_fb;
    l_mua = d_mua;
    l_mub = d_mub;*/
    l_2ndorder = d_2ndorder;
    l_moments = d_moments;
   
    //run simulation for multiple values of the system parameters
    if (l_moments) {
        long ridx = (idx/d_paths) % d_points;
        l_dx = d_dx[ridx];

        switch(d_domainx) {
            case 'a':
                l_amp = l_dx;
                break;
            case 'w':
                l_omega = l_dx;
                break;
            case 'f':
                l_force = l_dx;
                break;
            case 'g':
                l_gam = l_dx;
                break;
            case 'D':
                l_Dg = l_dx;
                break;
/*            case 'p':
                l_Dp = l_dx;
                if (l_mean != 0.0f) l_lambda = (l_mean*l_mean)/l_Dp;
                break;
            case 'l':
                l_lambda = l_dx;
                if (l_mean != 0.0f) l_Dp = (l_mean*l_mean)/l_lambda;
                break;
            case 'i':
                l_fa = l_dx;
                if (l_comp == 1) {
                    l_mua = -l_fa*l_mub/l_fb;
                    //l_fb = -l_fa*l_mub/l_mua;
                } else if (l_mean != 0.0f) {
                    l_mua = (l_fa - l_mean)*l_mub/(l_mean - l_fb);
                    //l_fb = (l_mean*(l_mua + l_mub) - l_fa*l_mub)/l_mua;
                }
                break;
            case 'j':
                l_fb = l_dx;
                if (l_comp == 1) {
                    l_mub = -l_fb*l_mua/l_fa;
                    //l_fa = -l_fb*l_mua/l_mub;
                } else if (l_mean != 0.0f) {
                    l_mub = (l_fb - l_mean)*l_mua/(l_mean - l_fa);
                    //l_fa = (l_mean*(l_mua + l_mub) - l_fb*l_mua)/l_mub;
                }
                break;
            case 'm':
                l_mua = l_dx;
                if (l_comp == 1) {
                    l_fa = -l_fb*l_mua/l_mub;
                    //l_mub = -l_fb*l_mua/l_fa;
                } else if (l_mean != 0.0f) {
                    l_fa = (l_mean*(l_mua + l_mub) - l_fb*l_mua)/l_mub;
                    //l_mub = (l_fb - l_mean)*l_mua/(l_mean - l_fa);
                }
                break;
            case 'n':
                l_mub = l_dx;
                if (l_comp == 1) {
                    l_fb = -l_fa*l_mub/l_mua;
                    //l_mua = -l_fa*l_mub/l_fb;
                } else if (l_mean != 0.0f) {
                    l_fb = (l_mean*(l_mua + l_mub) - l_fa*l_mub)/l_mua;
                    //l_mua = (l_fa - l_mean)*l_mub/(l_mean - l_fb);
                }
                break;*/
        }
    }

    //step size
    float l_dt, tmp;

    l_dt = 2.0f*PI/l_omega;

/*    if (l_lambda != 0.0f) {
        if (l_2ndorder) {
            tmp = 1.0f/l_lambda;
            if (tmp < l_dt) l_dt = tmp;
        }
    }*

    if (l_mua != 0.0f || l_mub != 0.0f) {
        float taua, taub;

        taua = 1.0f/l_mua;
        taub = 1.0f/l_mub;
        
        if (taua < taub) {
            tmp = taua;
        } else {
            tmp = taub;
        }

        if (tmp < l_dt) l_dt = tmp;
    }*/

    int l_spp;

    l_spp = d_spp;
    l_dt /= l_spp;

    //number of steps
    int l_samples;

    l_samples = d_samples;

/*    //jump countdowns
    int l_initnoise, l_pcd, l_dcd, l_dst;

    l_initnoise = d_initnoise;

    if (l_initnoise) {

        if (l_lambda != 0.0f) {
            if (l_2ndorder) {
                l_pcd = (int) floorf( -logf( hiprand_uniform(&l_state) )/l_lambda/l_dt + 0.5f );
            }
        }

        if (l_mua != 0.0f || l_mub != 0.0f) {
            float rn;
            rn = hiprand_uniform(&l_state);

            if (rn < 0.5f) {
                l_dst = 0;
                l_dcd = (int) floorf( -logf( hiprand_uniform(&l_state) )/l_mua/l_dt + 0.5f);
            } else {
                l_dst = 1;
                l_dcd = (int) floorf( -logf( hiprand_uniform(&l_state) )/l_mub/l_dt + 0.5f);
            }
        }

    } else {
        
        if (l_lambda != 0.0f) {
            if (l_2ndorder) {
                l_pcd = d_pcd[idx];
            }
        }
    
        if (l_mua != 0.0f || l_mub != 0.0f) {
            l_dcd = d_dcd[idx];
            l_dst = d_dst[idx];
        }*/

        int i, l_trigger;

        l_trigger = d_trigger;
    
        for (i = 0; i < l_samples; i++) {
            //algorithm
            if (l_2ndorder) {
                predcorr(l_x, l_x, l_v, l_v, l_w, l_w, l_pcd, l_pcd, &l_state, l_amp, l_omega, l_force, l_gam, l_Dg, l_2ndorder, l_Dp, l_lambda, l_comp, \
                         l_dcd, l_dcd, l_dst, l_dst, l_fa, l_fb, l_mua, l_mub, l_dt);
            } else {
                eulermaruyama(l_x, l_x, l_v, l_v, l_w, l_w, &l_state, l_amp, l_omega, l_force, l_gam, l_Dg, l_2ndorder, l_Dp, l_lambda, l_comp, \
                         l_dcd, l_dcd, l_dst, l_dst, l_fa, l_fb, l_mua, l_mub, l_dt);
            }
        
            if (l_moments) {
                if (l_trigger) {
                    l_sv1 += l_v1;
                    l_sv2 += l_v2;
                    l_sv12 += l_v1*l_v1;
                    l_sv22 += l_v2*l_v2;
                }
            }
        }
//    }

    //write back path parameters to the global memory
    d_x1[idx] = l_x1;
    d_x2[idx] = l_x2;
    d_v1[idx] = l_v1;
    d_v2[idx] = l_v2;
    d_w[idx] = l_w;
    d_sv1[idx] = l_sv1;
    d_sv2[idx] = l_sv2;
    d_sv12[idx] = l_sv12;
    d_sv22[idx] = l_sv22;
/*    d_pcd[idx] = l_pcd;
    d_dcd[idx] = l_dcd;
    d_dst[idx] = l_dst;*/
    d_states[idx] = l_state;
}

//prepare simulation
void prepare()
{
    //grid size
    h_paths = (h_paths/h_block)*h_block;
    h_threads = h_paths;

    if (h_moments) h_threads *= h_points;

    h_grid = h_threads/h_block;

    //number of steps
    h_steps = h_periods*h_spp;
     
    //host memory allocation
    size_f = h_threads*sizeof(float);
    size_i = h_threads*sizeof(int);
    size_ui = h_threads*sizeof(unsigned int);
    size_p = h_points*sizeof(float);

    h_x1 = (float*)malloc(size_f);
    h_x2 = (float*)malloc(size_f);
    h_fx1 = (float*)malloc(size_f);
    h_fx2 = (float*)malloc(size_f);
    h_v1 = (float*)malloc(size_f);
    h_v2 = (float*)malloc(size_f);
    h_w = (float*)malloc(size_f);
    h_fw = (float*)malloc(size_f);
    h_seeds = (unsigned int*)malloc(size_ui);

    //create & initialize host rng
    hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

    hiprandGenerate(gen, h_seeds, h_threads);
 
    //device memory allocation
    hipMalloc((void**)&d_x1, size_f);
    hipMalloc((void**)&d_x2, size_f);
    hipMalloc((void**)&d_fx1, size_f);
    hipMalloc((void**)&d_fx2, size_f);
    hipMalloc((void**)&d_v1, size_f);
    hipMalloc((void**)&d_v2, size_f);
    hipMalloc((void**)&d_w, size_f);
    hipMalloc((void**)&d_fw, size_f);
/*    hipMalloc((void**)&d_pcd, size_i);
    hipMalloc((void**)&d_dcd, size_i);
    hipMalloc((void**)&d_dst, size_i);*/
    hipMalloc((void**)&d_seeds, size_ui);
    hipMalloc((void**)&d_states, h_threads*sizeof(hiprandState));

    //copy seeds from host to device
    hipMemcpy(d_seeds, h_seeds, size_ui, hipMemcpyHostToDevice);

    //initialization of device rng
    init_dev_rng<<<h_grid, h_block>>>(d_seeds, d_states);

    free(h_seeds);
    hipFree(d_seeds);

    //moments specific requirements
    h_sv1 = (float*)malloc(size_f);
    h_sv2 = (float*)malloc(size_f);
    h_sv12 = (float*)malloc(size_f);
    h_sv22 = (float*)malloc(size_f);
    h_dx = (float*)malloc(size_p);

    float dxtmp = h_beginx;
    float dxstep = (h_endx - h_beginx)/h_points;

    int i;
        
    //set domainx
    for (i = 0; i < h_points; i++) {
        if (h_logx) {
            h_dx[i] = exp10f(dxtmp);
        } else {
            h_dx[i] = dxtmp;
        }
        dxtmp += dxstep;
    }
        
    hipMalloc((void**)&d_sv1, size_f);
    hipMalloc((void**)&d_sv2, size_f);
    hipMalloc((void**)&d_sv12, size_f);
    hipMalloc((void**)&d_sv22, size_f);
    hipMalloc((void**)&d_dx, size_p);
    
    hipMemcpy(d_dx, h_dx, size_p, hipMemcpyHostToDevice);
}

void copy_to_dev()
{
    hipMemcpy(d_x1, h_x1, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_x2, h_x2, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_fx1, h_fx1, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_fx2, h_fx2, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_v1, h_v1, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_v2, h_v2, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_fw, h_fw, size_f, hipMemcpyHostToDevice);

    hipMemcpy(d_sv1, h_sv1, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_sv2, h_sv2, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_sv12, h_sv12, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_sv22, h_sv22, size_f, hipMemcpyHostToDevice);
}

void copy_from_dev()
{
    hipMemcpy(h_x1, d_x1, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_x2, d_x2, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_fx1, d_fx1, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_fx2, d_fx2, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_v1, d_v1, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_v2, d_v2, size_f, hipMemcpyDeviceToHost);
}

//set initial conditions
void initial_conditions()
{
    hiprandGenerateUniform(gen, h_x1, h_threads); //x1 in (0,1]
    hiprandGenerateUniform(gen, h_x2, h_threads); //x2 in (0,1]
    hiprandGenerateUniform(gen, h_v1, h_threads);
    hiprandGenerateUniform(gen, h_v2, h_threads);
    hiprandGenerateUniform(gen, h_w, h_threads);

    long i;

    for (i = 0; i < h_threads; i++) {
        h_v1[i] = 4.0f*h_v1[i] - 2.0f; //v1 in (-2,2]
        h_v2[i] = 4.0f*h_v2[i] - 2.0f; //v2 in (-2,2]
        h_w[i] *= 2.0f*PI; //w in (0,2\pi]
    }

    memset(h_fx1, 0.0f, size_f);
    memset(h_fx2, 0.0f, size_f);
    memset(h_fw, 0.0f, size_f);
    
    memset(h_sv1, 0.0f, size_f);
    memset(h_sv2, 0.0f, size_f);
    memset(h_sv12, 0.0f, size_f);
    memset(h_sv22, 0.0f, size_f);

    copy_to_dev();
}

//calculate the first two moments of <v> and diffusion coefficient
void moments(float *av1, float *av2, float *av12, float *av22, float *dc1, float *dc2)
{
    float sv1, sv2, sv12, sv22, sx1, sx2, sx12, sx22, dt, tempo;//, tmp, taua, taub;
    int i, j;

    hipMemcpy(h_sv1, d_sv1, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_sv2, d_sv2, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_sv12, d_sv12, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_sv22, d_sv22, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_x1, d_x1, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_x2, d_x2, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_fx1, d_fx1, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_fx2, d_fx2, size_f, hipMemcpyDeviceToHost);

    unfold(h_x1, h_fx1);
    unfold(h_x2, h_fx2);

    for (j = 0; j < h_points; j++) {
        sv1 = 0.0f;
        sv2 = 0.0f;
        sv12 = 0.0f;
        sv22 = 0.0f;
        sx1 = 0.0f;
        sx2 = 0.0f;
        sx12 = 0.0f;
        sx22 = 0.0f;

        for (i = 0; i < h_paths; i++) {
            sv1 += h_sv1[j*h_paths + i];
            sv2 += h_sv2[j*h_paths + i];
            sv12 += h_sv12[j*h_paths + i];
            sv22 += h_sv22[j*h_paths + i];
            sx1 += h_x1[j*h_paths + i];
            sx2 += h_x2[j*h_paths + i];
            sx12 += h_x1[j*h_paths + i]*h_x1[j*h_paths + i];
            sx22 += h_x2[j*h_paths + i]*h_x2[j*h_paths + i];
        }

        av1[j] = sv1/((h_periods - h_trans)*h_spp)/h_paths;
        av2[j] = sv2/((h_periods - h_trans)*h_spp)/h_paths;
        av12[j] = sv12/((h_periods - h_trans)*h_spp)/h_paths;
        av22[j] = sv22/((h_periods - h_trans)*h_spp)/h_paths;

        //external driving
        if (h_domainx == 'w') {
            tempo = 2.0f*PI/h_dx[j];
        } else {
            tempo = 2.0f*PI/h_omega;
        }
       
        dt = tempo;

/*        //Poissonian
        if (h_lambda != 0.0f && h_2ndorder) {
            if (h_domainx == 'l') {
                tmp = 1.0f/h_dx[j];
            } else if (h_domainx == 'p' && h_mean != 0.0f) {
                tmp = 1.0f/(h_mean*h_mean/h_dx[j]);
            } else {
                tmp = 1.0f/h_lambda;
            }

            if (tmp < tempo) dt = tmp;
        }

        //Dichotomous
        if (h_mua != 0.0f || h_mub != 0.0f) {
            if (h_domainx == 'm') {
                taua = 1.0f/h_dx[j];
                taub = 1.0f/h_mub;

                if (h_comp) {
                    tmp = 1.0f/(-h_fb*h_dx[j]/h_fa);
                } else if (h_mean != 0.0f) {
                    tmp = 1.0f/((h_fb - h_mean)*h_dx[j]/(h_mean - h_fa));
                } else {
                    tmp = taub;
                //}
            
                if (taua <= tmp) {
                    if (taua < tempo) dt = taua;
                } else {
                    if (tmp < tempo) dt = tmp;
                }
            } else if (h_domainx == 'n') {
                taua = 1.0f/h_mua;
                taub = 1.0f/h_dx[j];

                if (h_comp) {
                    tmp = 1.0f/(-h_fa*h_dx[j]/h_fb);
                } else if (h_mean != 0.0f) {
                    tmp = 1.0f/((h_fa - h_mean)*h_dx[j]/(h_mean - h_fb));
                } else {
                    tmp = taua;
                //}

                if (taub <= tmp) {
                    if (taub < tempo) dt = taub;
                } else {
                    if (tmp < tempo) dt = tmp;
                }
            } else if (h_domainx == 'i') {
                taua = 1.0f/h_mua;
                taub = 1.0f/h_mub;

                if (h_comp) {
                    tmp = 1.0f/(-h_dx[j]*h_mub/h_fb);
                } else if (h_mean != 0.0f) {
                    tmp = 1.0f/((h_dx[j] - h_mean)*h_mub/(h_mean - h_fb));
                } else {
                    tmp = taua;
                }

                if (taub <= tmp) {
                    if (taub < tempo) dt = taub;
                } else {
                    if (tmp < tempo) dt = tmp;
                }
            } else if (h_domainx == 'j') {
                taua = 1.0f/h_mua;
                taub = 1.0f/h_mub;

                if (h_comp) {
                    tmp = 1.0f/(-h_dx[j]*h_mua/h_fa);
                } else if (h_mean != 0.0f) {
                    tmp = 1.0f/((h_dx[j] - h_mean)*h_mua/(h_mean - h_fa));
                } else {
                    tmp = taub;
                }

                if (taua <= tmp) {
                    if (taua < tempo) dt = taua;
                } else {
                    if (tmp < tempo) dt = tmp;
                }
            } else {
                taua = 1.0f/h_mua;
                taub = 1.0f/h_mub;

                if (taua < taub) {
                    if (taua < tempo) dt = taua;
                } else {
                    if (taub < tempo) dt = taub;
                }
            }
        }*/

        dt /= h_spp;

        sx1 /= h_paths;
        sx2 /= h_paths;
        sx12 /= h_paths;
        sx22 /= h_paths;
        dc1[j] = (sx12 - sx1*sx1)/(2.0f*h_steps*dt);
        dc2[j] = (sx22 - sx2*sx2)/(2.0f*h_steps*dt);
    }
}

//calculate ensemble average
void ensemble_average(float *h_x1, *float h_x2, float *h_v1, float *h_v2, float &sx1, float &sx2, float &sx12, float &sx22, float &sv1, float &sv2, float &sv12, float &sv22)
{
    long i;

    sx1 = 0.0f;
    sx2 = 0.0f;
    sv1 = 0.0f;
    sv2 = 0.0f;
    sx12 = 0.0f;
    sx22 = 0.0f;
    sv12 = 0.0f;
    sv22 = 0.0f;

    for (i = 0; i < h_threads; i++) {
        sx1 += h_x1[i];
        sx2 += h_x2[i];
        sv1 += h_v1[i];
        sv2 += h_v2[i];
        sx12 += h_x1[i]*h_x1[i];
        sx22 += h_x2[i]*h_x2[i];
        sv12 += h_v1[i]*h_v1[i];
        sv22 += h_v2[i]*h_v2[i];
    }

    sx1 /= h_threads;
    sx2 /= h_threads;
    sv1 /= h_threads;
    sv2 /= h_threads;
    sx12 /= h_threads;
    sx22 /= h_threads;
    sv12 /= h_threads;
    sv22 /= h_threads;
}

//free memory
void finish()
{
    free(h_x1);
    free(h_x2);
    free(h_fx1);
    free(h_fx2);
    free(h_v1);
    free(h_v2);
    free(h_w);
    free(h_fw);
    
    hiprandDestroyGenerator(gen);
    hipFree(d_x1);
    hipFree(d_x2);
    hipFree(d_fx1);
    hipFree(d_fx2);
    hipFree(d_v1);
    hipFree(d_v2);
    hipFree(d_w);
    hipFree(d_fw);
/*    hipFree(d_pcd);
    hipFree(d_dcd);
    hipFree(d_dst);*/
    hipFree(d_states);
    
    free(h_sv1);
    free(h_sv2);
    free(h_sv12);
    free(h_sv22);
    free(h_dx);

    hipFree(d_sv1);
    hipFree(d_sv2);
    hipFree(d_sv12);
    hipFree(d_sv22);
    hipFree(d_dx);
}

int main(int argc, char **argv)
{
    parse_cla(argc, argv);
    if (!h_moments && !h_traj && !h_hist) {
        usage(argv);
        return -1;
    }

    prepare();

    initial_conditions();

/*    h_initnoise = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(d_initnoise), &h_initnoise, sizeof(int));

    if ( (h_lambda != 0.0f && h_2ndorder) || (h_mua != 0.0f || h_mub != 0.0f) ) {
        h_initnoise = 1;
        hipMemcpyToSymbol(HIP_SYMBOL(d_initnoise), &h_initnoise, sizeof(int));

        run_sim<<<h_grid, h_block>>>(d_x, d_v, d_w, d_sv, d_sv2, d_dx, d_pcd, d_dcd, d_dst, d_states);

        h_initnoise = 0;
        hipMemcpyToSymbol(HIP_SYMBOL(d_initnoise), &h_initnoise, sizeof(int));
    }*/

    //asymptotic long time average velocity <<v>>, <<v^2>> and diffusion coefficient
    if (h_moments) {
        float *av1, *av2, *av12, *av22, *dc1, *dc2;
        long i;

        av1 = (float*)malloc(size_p);
        av2 = (float*)malloc(size_p);
        av12 = (float*)malloc(size_p);
        av22 = (float*)malloc(size_p);
        dc1 = (float*)malloc(size_p);
        dc2 = (float*)malloc(size_p);

        if ( !strcmp(h_domain, "1d") ) { 

            h_trigger = 0;
            hipMemcpyToSymbol(HIP_SYMBOL(d_trigger), &h_trigger, sizeof(int));

            for (i = 0; i < h_steps; i += h_samples) {
                run_sim<<<h_grid, h_block>>>(d_x1, d_x2, d_v1, d_v2, d_w, d_sv1, d_sv2, d_sv12, d_sv22, d_dx, d_states);//d_pcd, d_dcd, d_dst, d_states);
                fold<<<h_grid, h_block>>>(d_x1, d_fx1, 1.0f);
                fold<<<h_grid, h_block>>>(d_x2, d_fx2, 1.0f);
                fold<<<h_grid, h_block>>>(d_w, d_fw, (2.0f*PI));
                if (i == h_trans*h_spp) {
                    h_trigger = 1;
                    hipMemcpyToSymbol(HIP_SYMBOL(d_trigger), &h_trigger, sizeof(int));
                }
            }

            moments(av1, av2, av12, av22, dc1, dc2);

            printf("#%c <<v1>> <<v1^2>> <<v2>> <<v2^2>> D_x1 D_x2\n", h_domainx);
            for (i = 0; i < h_points; i++) {
                printf("%e %e %e %e %e %e %e\n", h_dx[i], av1[i], av12[i], av2[i], av22[i], dc1[i], dc2[i]);
            }

        } else {
            float h_dy, dytmp, dystep;
            int j, k;
            
            dytmp = h_beginy;
            dystep = (h_endy - h_beginy)/h_points;
            
            printf("#%c %c <<v1>> <<v1^2>> <<v2>> <<v2^2>> D_x1 D_x2\n", h_domainx, h_domainy);
            
            for (k = 0; k < h_points; k++) {
                if (h_logy) {
                    h_dy = exp10f(dytmp);
                } else {
                    h_dy = dytmp;
                }

                switch(h_domainy) {
                    case 'a':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_amp), &h_dy, sizeof(float));
                        break;
                    case 'w':
                        h_omega = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_omega), &h_omega, sizeof(float));
                        break;
                    case 'f':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_force), &h_dy, sizeof(float));
                        break;
                    case 'g':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_gam), &h_dy, sizeof(float));
                        break;
                    case 'D':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_Dg), &h_dy, sizeof(float));
                        break;
/*                    case 'p':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_Dp), &h_dy, sizeof(float));
                        break;
                    case 'l':
                        h_lambda = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_lambda), &h_lambda, sizeof(float));
                        break;
                    case 'i':
                        h_fa = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_fa), &h_fa, sizeof(float));
                        break;
                    case 'j':
                        h_fb = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_fb), &h_fb, sizeof(float));
                        break;
                    case 'm':
                        h_mua = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_mua), &h_mua, sizeof(float));
                        break;
                    case 'n':
                        h_mub = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_mub), &h_mub, sizeof(float));
                        break;*/
                }
 
                h_trigger = 0;
                hipMemcpyToSymbol(HIP_SYMBOL(d_trigger), &h_trigger, sizeof(int));

                for (i = 0; i < h_steps; i += h_samples) {
                    run_sim<<<h_grid, h_block>>>(d_x1, d_x2, d_v1, d_v2 d_w, d_sv1, d_sv2, d_sv12, d_sv22, d_dx, d_states);//d_pcd, d_dcd, d_dst, d_states);
                    fold<<<h_grid, h_block>>>(d_x1, d_fx1, 1.0f);
                    fold<<<h_grid, h_block>>>(d_x2, d_fx2, 1.0f);
                    fold<<<h_grid, h_block>>>(d_w, d_fw, (2.0f*PI));
                    if (i == h_trans*h_spp) {
                        h_trigger = 1;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_trigger), &h_trigger, sizeof(int));
                    }
                }
 
                moments(av1, av2, av12, av22, dc1, dc2);
                
                for (j = 0; j < h_points; j++) {
                    printf("%e %e %e %e %e %e %e %e\n", h_dx[j], h_dy, av1[j], av12[j], av2[j], av22[j], dc1[j], dc2[j]);
                }

                //blank line for plotting purposes
                printf("\n");

                initial_conditions();

/*                if ( (h_lambda != 0.0f && h_2ndorder) || (h_mua != 0.0f || h_mub != 0.0f) ) {
                    h_initnoise = 1;
                    hipMemcpyToSymbol(HIP_SYMBOL(d_initnoise), &h_initnoise, sizeof(int));

                    run_sim<<<h_grid, h_block>>>(d_x, d_v, d_w, d_sv, d_sv2, d_dx, d_pcd, d_dcd, d_dst, d_states);

                    h_initnoise = 0;
                    hipMemcpyToSymbol(HIP_SYMBOL(d_initnoise), &h_initnoise, sizeof(int));
                }*/

                dytmp += dystep;
           }
        }

        free(av1);
        free(av2);
        free(av12);
        free(av22);
        free(dc1);
        free(dc2);
    }

    //ensemble averaged trajectory <x>(t), <v>(t) and <x^2>(t), <v^2>(t)
    if (h_traj) {
        float t, sx1, sx2, sv1, sv2, sx12, sx22, sv12, sv22, dt, tmp;//, taua, taub;
        long i;

        dt = 2.0f*PI/h_omega;
        tmp = dt;

/*        if (h_lambda != 0.0f && h_2ndorder) tmp = 1.0f/h_lambda;

        if (h_mua != 0.0f || h_mub != 0.0f) {
            taua = 1.0f/h_mua;
            taub = 1.0f/h_mub;

            if (taua < taub) {
                tmp = taua;
            } else {
                tmp = taub;
            }
        }

        if (tmp < dt) dt = tmp;*/

        dt /= h_spp;

        printf("#t <x1> <v1> <x1^2> <v1^2> <x2> <v2> <x2^2> <v2^2>\n");
        
        for (i = 0; i < h_steps; i += h_samples) {
            run_sim<<<h_grid, h_block>>>(d_x1, d_x2, d_v1, d_v2, d_w, d_sv1, d_sv2, d_sv12, d_sv22, d_dx, d_states);//d_pcd, d_dcd, d_dst, d_states);
            copy_from_dev();
            unfold(h_x1, h_fx1);
            unfold(h_x2, h_fx2);
            t = i*dt;
            ensemble_average(h_x1, h_x2, h_v1, h_v2, sx1, sx2, sx12, sx22, sv1, sv2, sv12, sv22);
            printf("%e %e %e %e %e %e %e %e %e\n", t, sx1, sv1, sx12, sv12, sx2, sv2, sx22, sv22);
            fold<<<h_grid, h_block>>>(d_x1, d_fx1, 1.0f);
            fold<<<h_grid, h_block>>>(d_x2, d_fx2, 1.0f);
            fold<<<h_grid, h_block>>>(d_w, d_fw, (2.0f*PI));
        }
    }

    //the final position x and velocity v of all paths
    if (h_hist) {
        long i;

        for (i = 0; i < h_steps; i += h_samples) {
            run_sim<<<h_grid, h_block>>>(d_x1, d_x2, d_v1, d_v2, d_w, d_sv1, d_sv2, d_sv12, d_sv22, d_dx, d_states);//d_pcd, d_dcd, d_dst, d_states);
            fold<<<h_grid, h_block>>>(d_x1, d_fx1, 1.0f);
            fold<<<h_grid, h_block>>>(d_x2, d_fx2, 1.0f);
            fold<<<h_grid, h_block>>>(d_w, d_fw, (2.0f*PI));
        }
        
        copy_from_dev();

        printf("#x1 v1 x2 v2\n");
        
        for (i = 0; i < h_threads; i++) {
            printf("%e %e %e %e\n", h_x1[i], h_v1[i], h_x2[i], h_v2[i]); 
        }
    }

    finish();

    return 0;
}
